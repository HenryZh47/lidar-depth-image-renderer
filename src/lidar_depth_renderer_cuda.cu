#include "hip/hip_runtime.h"
#include "lidar_depth_renderer/lidar_depth_renderer_cuda.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

/* -------------------- CUDA Util Functions ------------------------*/
#define CHECK_GPU(ans) \
  { gpu_assert((ans), __FILE__, __LINE__); }
#define CHECK_GPU_SAFE(ans) \
  { gpu_assert((ans), __FILE__, __LINE__, false); }
inline void gpu_assert(hipError_t code, const char* file, int line,
                       bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort) {
      exit(code);
    }
  }
}

void *cuda_malloc(const size_t size) {
  void *cloud_ptr;
  CHECK_GPU(hipMalloc(&cloud_ptr, size));
  return cloud_ptr;
}

void cuda_memcpy_to_dev(void *dst, const void *src, const size_t size) {
  CHECK_GPU(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
}

void cuda_free(void *dst) {
  CHECK_GPU(hipFree(dst));
}

/* -------------------- CUDA Structures Functions ------------------------*/
// make device available camera info
struct CudaCameraInfo {
  int width;
  int height;

  float fx;
  float fy;
  float cx;
  float cy;

  CudaCameraInfo(const sensor_msgs::CameraInfo &info)
      : width(info.width),
        height(info.height),
        fx(info.K[0]),
        fy(info.K[4]),
        cx(info.K[2]),
        cy(info.K[5]) {}
}; // Struct CudaCameraInfo

// Need to have padding to correctly interpret PCL PointXYZ
struct CudaPoint {
  float x;
  float y;
  float z;
  float padding;
}; // Struct CudaPoint

// make device available transform info
struct CudaTransform {
  float rot[3][3];
  float trans[3];

  CudaTransform(const tf2::Transform &tf) {
    const auto &basis = tf.getBasis();
    const auto &origin = tf.getOrigin();
    for (int i = 0; i < 3; i++) {
      trans[i] = origin[i];
      for (int j = 0; j < 3; j++) {
        rot[i][j] = basis[i][j];
      }
    }
  }

}; // Struct CudaTransform

/* -------------------- CUDA Kernel Functions ------------------------*/
// stores result to image_scratch_buf
// image_scratch_buf should be set to all 255
__global__ void kernel_render(const CudaPoint *cloud_points,
                              const size_t num_points,
                              const CudaCameraInfo camera_info,
                              const CudaTransform to_camera_tf,
                              const int bloat_factor,
                              uint32_t *image_scratch_buf) {
  // get point index
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx > num_points) return;
  const auto point = cloud_points[idx];

  // transform point
  const auto &trans = to_camera_tf.trans;
  const auto &rot = to_camera_tf.rot;

  const float x = rot[0][0] * point.x +
                  rot[0][1] * point.y +
                  rot[0][2] * point.z + trans[0];
  const float y = rot[1][0] * point.x +
                  rot[1][1] * point.y +
                  rot[1][2] * point.z + trans[1];
  const float z = rot[2][0] * point.x +
                  rot[2][1] * point.y +
                  rot[2][2] * point.z + trans[2];
  if (z <= 0) return;
  
  // project point
  const auto fx = camera_info.fx;
  const auto fy = camera_info.fy;
  const auto cx = camera_info.cx;
  const auto cy = camera_info.cy;

  const auto u = static_cast<int>((fx * x) / z + cx);
  const auto v = static_cast<int>((fy * y) / z + cy);

  // render point
  const auto width = camera_info.width;
  const auto height = camera_info.height;

  if (u - bloat_factor >= 0 && u + bloat_factor < width &&
      v - bloat_factor >= 0 && v + bloat_factor < height) {
    auto cur_depth = static_cast<uint32_t>(z * 2.0);
    // cap to 254
    cur_depth = cur_depth > 255 ? 254 : cur_depth;
    for (int i = -bloat_factor; i <= bloat_factor; i++) {
      for (int j = -bloat_factor; j <= bloat_factor; j++) {
        // convert to 1d buffer
        const auto buf_index = (v + i) * width + u;
        // atomic min to buffer
        atomicMin(image_scratch_buf + buf_index, cur_depth);
      }
    }
  }
}

__global__ void kernel_populate_image_buf(uint32_t *image_scratch_buf,
                                          uint8_t *image_buf,
                                          const size_t num_pixels) {
  const auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_pixels) return;

  const auto scratch_val = image_scratch_buf[idx];
  // convert to 0 if scratch_val is MAX_UINT32_T
  image_buf[idx] = scratch_val == 0xFFFFFFFFu ? 0 : static_cast<uint8_t>(scratch_val);
}

/* -------------------- Renderer Functions ------------------------*/
void LidarDepthRendererCuda::init(int height, int width) {
  if (height < 0 || width < 0) return;
  image_height = height;
  image_width = width;
  
  // need to cuda malloc image buffer and image scratch buffer
  init_image_buf();
}

void LidarDepthRendererCuda::render(cv::Mat &result,
                                    const sensor_msgs::CameraInfo camera_info,
                                    const tf2::Transform &to_camera_tf,
                                    const int bloat_factor) {
  // clear scratch buffer
  set_image_buf();

  // render for each point cloud
  const dim3 threads_per_block(256);
  for (const auto &cloud : *cloud_ptr) {
    // get number of thread blocks
    const auto num_points = cloud.second;
    const dim3 num_blocks((num_points / threads_per_block.x) + 1);
    kernel_render<<<num_blocks, threads_per_block>>>(reinterpret_cast<const CudaPoint*>(cloud.first), 
                                                     cloud.second,
                                                     CudaCameraInfo(camera_info),
                                                     CudaTransform(to_camera_tf),
                                                     bloat_factor,
                                                     image_scratch_buf);
    CHECK_GPU(hipPeekAtLastError());
  }

  // populate final image buffer
  const auto num_pixels = image_width * image_height;
  const dim3 num_blocks(num_pixels / threads_per_block.x);
  kernel_populate_image_buf<<<num_blocks, threads_per_block>>>(image_scratch_buf,
                                                               image_buf,
                                                               num_pixels);
  
  // populate result cv mat
  CHECK_GPU(hipMemcpy(result.data, image_buf,
                       image_width * image_height * sizeof(uint8_t),
                       hipMemcpyDeviceToHost));
}

void LidarDepthRendererCuda::set_cloud(const CloudWindowPtr new_cloud_ptr) {
  cloud_ptr = new_cloud_ptr;
}

// CUDA implementation has enum 2
int LidarDepthRendererCuda::query_implementation() {
  return 2;
}

void LidarDepthRendererCuda::init_image_buf() {
  int num_pixels = image_height * image_width;
  // reallocate depth_image and scratch_space on device memory
  cuda_free(image_scratch_buf);
  cuda_free(image_buf);
  image_scratch_buf = reinterpret_cast<uint32_t*>(cuda_malloc(num_pixels * sizeof(uint32_t)));
  image_buf = reinterpret_cast<uint8_t*>(cuda_malloc(num_pixels * sizeof(uint8_t)));
  // reset image_scratch_buf
  set_image_buf();
}

void LidarDepthRendererCuda::set_image_buf() {
  int num_pixels = image_height * image_width;
  CHECK_GPU(hipMemset(image_scratch_buf, 0xFF, num_pixels * sizeof(uint32_t)));
}
